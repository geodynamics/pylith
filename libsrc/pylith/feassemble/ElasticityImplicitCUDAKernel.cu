
#include <hip/hip_runtime.h>
#include <stdexcept>

#define CHECK_CUDA_ERROR_MSG(e, msg) do {if (e) return e;} while (0)

__global__ void integrateElasticity(float *elemMat, float *geometry, float *analytic)
{
  const int        gridIdx = blockIdx.x + blockIdx.y*gridDim.x; /* Indexes element batch */
  const int        Kidx    = threadIdx.x + threadIdx.y*12; /* This is (i,j) for test and basis functions */
  const int        idx     = Kidx;                        /* Unique thread ID (K block is for a single element) */

  const int        Goffset = gridIdx*288;
  __shared__ float G[288];
  const int        Koffset = Kidx*9;
  float            K[9];
  const int        Eoffset = gridIdx*4608;
  float            E       = 0.0;

  // Load geometry from global memory into G in shared memory
G[idx+0] = geometry[Goffset+idx+0];
G[idx+144] = geometry[Goffset+idx+144];

  /* Copy K^{ij} into local memory (not coalesced) */
  K[0] = analytic[Koffset+0];
  K[1] = analytic[Koffset+1];
  K[2] = analytic[Koffset+2];
  K[3] = analytic[Koffset+3];
  K[4] = analytic[Koffset+4];
  K[5] = analytic[Koffset+5];
  K[6] = analytic[Koffset+6];
  K[7] = analytic[Koffset+7];
  K[8] = analytic[Koffset+8];
  __syncthreads(); /* Make G available */
  /* Do contraction */ 
  /*   NEED TO INTERLEAVE CONTRACTIONS OF CONCURRENT ELEMENTS? See Volkov talk */
  E += G[0] * K[0];
E += G[1] * K[1];
E += G[2] * K[2];
E += G[3] * K[3];
E += G[4] * K[4];
E += G[5] * K[5];
E += G[6] * K[6];
E += G[7] * K[7];
E += G[8] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+0] = E;
  E = 0.0;
  E += G[9] * K[0];
E += G[10] * K[1];
E += G[11] * K[2];
E += G[12] * K[3];
E += G[13] * K[4];
E += G[14] * K[5];
E += G[15] * K[6];
E += G[16] * K[7];
E += G[17] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+144] = E;
  E = 0.0;
  E += G[18] * K[0];
E += G[19] * K[1];
E += G[20] * K[2];
E += G[21] * K[3];
E += G[22] * K[4];
E += G[23] * K[5];
E += G[24] * K[6];
E += G[25] * K[7];
E += G[26] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+288] = E;
  E = 0.0;
  E += G[27] * K[0];
E += G[28] * K[1];
E += G[29] * K[2];
E += G[30] * K[3];
E += G[31] * K[4];
E += G[32] * K[5];
E += G[33] * K[6];
E += G[34] * K[7];
E += G[35] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+432] = E;
  E = 0.0;
  E += G[36] * K[0];
E += G[37] * K[1];
E += G[38] * K[2];
E += G[39] * K[3];
E += G[40] * K[4];
E += G[41] * K[5];
E += G[42] * K[6];
E += G[43] * K[7];
E += G[44] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+576] = E;
  E = 0.0;
  E += G[45] * K[0];
E += G[46] * K[1];
E += G[47] * K[2];
E += G[48] * K[3];
E += G[49] * K[4];
E += G[50] * K[5];
E += G[51] * K[6];
E += G[52] * K[7];
E += G[53] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+720] = E;
  E = 0.0;
  E += G[54] * K[0];
E += G[55] * K[1];
E += G[56] * K[2];
E += G[57] * K[3];
E += G[58] * K[4];
E += G[59] * K[5];
E += G[60] * K[6];
E += G[61] * K[7];
E += G[62] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+864] = E;
  E = 0.0;
  E += G[63] * K[0];
E += G[64] * K[1];
E += G[65] * K[2];
E += G[66] * K[3];
E += G[67] * K[4];
E += G[68] * K[5];
E += G[69] * K[6];
E += G[70] * K[7];
E += G[71] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1008] = E;
  E = 0.0;
  E += G[72] * K[0];
E += G[73] * K[1];
E += G[74] * K[2];
E += G[75] * K[3];
E += G[76] * K[4];
E += G[77] * K[5];
E += G[78] * K[6];
E += G[79] * K[7];
E += G[80] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1152] = E;
  E = 0.0;
  E += G[81] * K[0];
E += G[82] * K[1];
E += G[83] * K[2];
E += G[84] * K[3];
E += G[85] * K[4];
E += G[86] * K[5];
E += G[87] * K[6];
E += G[88] * K[7];
E += G[89] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1296] = E;
  E = 0.0;
  E += G[90] * K[0];
E += G[91] * K[1];
E += G[92] * K[2];
E += G[93] * K[3];
E += G[94] * K[4];
E += G[95] * K[5];
E += G[96] * K[6];
E += G[97] * K[7];
E += G[98] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1440] = E;
  E = 0.0;
  E += G[99] * K[0];
E += G[100] * K[1];
E += G[101] * K[2];
E += G[102] * K[3];
E += G[103] * K[4];
E += G[104] * K[5];
E += G[105] * K[6];
E += G[106] * K[7];
E += G[107] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1584] = E;
  E = 0.0;
  E += G[108] * K[0];
E += G[109] * K[1];
E += G[110] * K[2];
E += G[111] * K[3];
E += G[112] * K[4];
E += G[113] * K[5];
E += G[114] * K[6];
E += G[115] * K[7];
E += G[116] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1728] = E;
  E = 0.0;
  E += G[117] * K[0];
E += G[118] * K[1];
E += G[119] * K[2];
E += G[120] * K[3];
E += G[121] * K[4];
E += G[122] * K[5];
E += G[123] * K[6];
E += G[124] * K[7];
E += G[125] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+1872] = E;
  E = 0.0;
  E += G[126] * K[0];
E += G[127] * K[1];
E += G[128] * K[2];
E += G[129] * K[3];
E += G[130] * K[4];
E += G[131] * K[5];
E += G[132] * K[6];
E += G[133] * K[7];
E += G[134] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2016] = E;
  E = 0.0;
  E += G[135] * K[0];
E += G[136] * K[1];
E += G[137] * K[2];
E += G[138] * K[3];
E += G[139] * K[4];
E += G[140] * K[5];
E += G[141] * K[6];
E += G[142] * K[7];
E += G[143] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2160] = E;
  E = 0.0;
  E += G[144] * K[0];
E += G[145] * K[1];
E += G[146] * K[2];
E += G[147] * K[3];
E += G[148] * K[4];
E += G[149] * K[5];
E += G[150] * K[6];
E += G[151] * K[7];
E += G[152] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2304] = E;
  E = 0.0;
  E += G[153] * K[0];
E += G[154] * K[1];
E += G[155] * K[2];
E += G[156] * K[3];
E += G[157] * K[4];
E += G[158] * K[5];
E += G[159] * K[6];
E += G[160] * K[7];
E += G[161] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2448] = E;
  E = 0.0;
  E += G[162] * K[0];
E += G[163] * K[1];
E += G[164] * K[2];
E += G[165] * K[3];
E += G[166] * K[4];
E += G[167] * K[5];
E += G[168] * K[6];
E += G[169] * K[7];
E += G[170] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2592] = E;
  E = 0.0;
  E += G[171] * K[0];
E += G[172] * K[1];
E += G[173] * K[2];
E += G[174] * K[3];
E += G[175] * K[4];
E += G[176] * K[5];
E += G[177] * K[6];
E += G[178] * K[7];
E += G[179] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2736] = E;
  E = 0.0;
  E += G[180] * K[0];
E += G[181] * K[1];
E += G[182] * K[2];
E += G[183] * K[3];
E += G[184] * K[4];
E += G[185] * K[5];
E += G[186] * K[6];
E += G[187] * K[7];
E += G[188] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+2880] = E;
  E = 0.0;
  E += G[189] * K[0];
E += G[190] * K[1];
E += G[191] * K[2];
E += G[192] * K[3];
E += G[193] * K[4];
E += G[194] * K[5];
E += G[195] * K[6];
E += G[196] * K[7];
E += G[197] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3024] = E;
  E = 0.0;
  E += G[198] * K[0];
E += G[199] * K[1];
E += G[200] * K[2];
E += G[201] * K[3];
E += G[202] * K[4];
E += G[203] * K[5];
E += G[204] * K[6];
E += G[205] * K[7];
E += G[206] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3168] = E;
  E = 0.0;
  E += G[207] * K[0];
E += G[208] * K[1];
E += G[209] * K[2];
E += G[210] * K[3];
E += G[211] * K[4];
E += G[212] * K[5];
E += G[213] * K[6];
E += G[214] * K[7];
E += G[215] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3312] = E;
  E = 0.0;
  E += G[216] * K[0];
E += G[217] * K[1];
E += G[218] * K[2];
E += G[219] * K[3];
E += G[220] * K[4];
E += G[221] * K[5];
E += G[222] * K[6];
E += G[223] * K[7];
E += G[224] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3456] = E;
  E = 0.0;
  E += G[225] * K[0];
E += G[226] * K[1];
E += G[227] * K[2];
E += G[228] * K[3];
E += G[229] * K[4];
E += G[230] * K[5];
E += G[231] * K[6];
E += G[232] * K[7];
E += G[233] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3600] = E;
  E = 0.0;
  E += G[234] * K[0];
E += G[235] * K[1];
E += G[236] * K[2];
E += G[237] * K[3];
E += G[238] * K[4];
E += G[239] * K[5];
E += G[240] * K[6];
E += G[241] * K[7];
E += G[242] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3744] = E;
  E = 0.0;
  E += G[243] * K[0];
E += G[244] * K[1];
E += G[245] * K[2];
E += G[246] * K[3];
E += G[247] * K[4];
E += G[248] * K[5];
E += G[249] * K[6];
E += G[250] * K[7];
E += G[251] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+3888] = E;
  E = 0.0;
  E += G[252] * K[0];
E += G[253] * K[1];
E += G[254] * K[2];
E += G[255] * K[3];
E += G[256] * K[4];
E += G[257] * K[5];
E += G[258] * K[6];
E += G[259] * K[7];
E += G[260] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+4032] = E;
  E = 0.0;
  E += G[261] * K[0];
E += G[262] * K[1];
E += G[263] * K[2];
E += G[264] * K[3];
E += G[265] * K[4];
E += G[266] * K[5];
E += G[267] * K[6];
E += G[268] * K[7];
E += G[269] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+4176] = E;
  E = 0.0;
  E += G[270] * K[0];
E += G[271] * K[1];
E += G[272] * K[2];
E += G[273] * K[3];
E += G[274] * K[4];
E += G[275] * K[5];
E += G[276] * K[6];
E += G[277] * K[7];
E += G[278] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+4320] = E;
  E = 0.0;
  E += G[279] * K[0];
E += G[280] * K[1];
E += G[281] * K[2];
E += G[282] * K[3];
E += G[283] * K[4];
E += G[284] * K[5];
E += G[285] * K[6];
E += G[286] * K[7];
E += G[287] * K[8];

  /* Store contraction result */
  elemMat[Eoffset+idx+4464] = E;
  
  
}

hipError_t setupKernel(const int dim, const int numBasisFuncs, const int numCells, float *K, float **geometry, float **elemMat, float **analytic_gpu, float **geometry_gpu, float **elemMat_gpu)
{
  const int   N             = numCells;
  const int   numComponents = dim;
  int         Ksize         = (numBasisFuncs*numComponents * dim)*(numBasisFuncs*numComponents * dim);
  size_t      Kbytes        = Ksize * sizeof(float);
  int         Gsize         = N*dim*dim;
  size_t      Gbytes        = Gsize * sizeof(float);
  int         Esize         = (numBasisFuncs*numComponents)*(numBasisFuncs*numComponents);
  size_t      Ebytes        = Esize * sizeof(float);
  hipError_t cerr;

  cerr = hipMalloc(analytic_gpu, Kbytes);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Allocation failure");
  cerr = hipMemcpy(*analytic_gpu, K, Kbytes, hipMemcpyHostToDevice);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Allocation failure");
  cerr = hipHostMalloc(geometry, Gbytes, hipHostMallocDefault, hipHostMallocDefault);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Allocation failure");
  cerr = hipMalloc(geometry_gpu, Gbytes);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Allocation failure");
  cerr = hipHostMalloc(elemMat, Ebytes, hipHostMallocDefault, hipHostMallocDefault);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Allocation failure");
  cerr = hipMalloc(elemMat_gpu, Ebytes);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Allocation failure");
  for(int i = 0; i < Esize; ++i) {(*elemMat)[i] = 0.0;}
  return hipSuccess;
}

// Calculate a conforming thread grid for N kernels
void calculateGrid(const int N, const int blockSize, unsigned int& x, unsigned int& y, unsigned int& z)
{
  z = 1;
  if (N % blockSize) {
    // 'Invalid block size '+str(blockSize)+' for '+str(N)+' elements'
    throw std::runtime_error("Invalid block size");
  }
  const int Nblocks = N/blockSize;
  for(x = (int) (sqrt(Nblocks) + 0.5); x > 0; --x) {
    y = Nblocks/x;
    if (x*y == Nblocks) break;
  }
  if (x*y != Nblocks) {
    // 'Could not find partition for '+str(N)+' with block size '+str(blockSize)
    throw std::runtime_error("Could not find partition");
  }
  return;
}

hipError_t launchKernel(const int spaceDim, const int numBasis, const int elementBatchSize, const int numConcurrentElements,
                         const int N, float *analytic_gpu, float *geometry_gpu, float *elemMat_gpu)
{
  dim3 grid, block;
  block.x = numBasis*spaceDim;
  block.y = numBasis*spaceDim;
  block.z = numConcurrentElements;
  calculateGrid(N, elementBatchSize, grid.x, grid.y, grid.z);
  // self.logPrint('Running %d elements with Thread Block size %d' % (N, reduce(int.__mul__, blockDim)), debugLevel=1, debugSection=self.section)
  // self.logPrint('  using grid '+str(self.calculateGrid(N, elementBatchSize)), debugLevel=1, debugSection=self.section)
  integrateElasticity<<<grid, block>>>(elemMat_gpu, geometry_gpu, analytic_gpu);
  return hipSuccess;
}

hipError_t cleanupKernel(float *geometry, float *elemMat, float *analytic_gpu, float *geometry_gpu, float *elemMat_gpu)
{
  hipError_t cerr;

  cerr = hipFree(analytic_gpu);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Deallocation failure");
  cerr = hipFree(geometry_gpu);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Deallocation failure");
  cerr = hipFree(elemMat_gpu);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Deallocation failure");
  cerr = hipHostFree(geometry);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Deallocation failure");
  cerr = hipHostFree(elemMat);CHECK_CUDA_ERROR_MSG(cerr, "CUDA Deallocation failure");
  return hipSuccess;
}
